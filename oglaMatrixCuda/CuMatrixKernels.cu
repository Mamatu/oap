#include "hip/hip_runtime.h"
#include "CuMatrixProcedures.h"
#define DEBUG

extern "C" __device__ void CUDA_PrintInfo(MatrixStructure* ms) {
#ifdef DEBUG
    printf("KERNELS ms == %llu %llu,%llu,%llu,%llu \n", ms, ms->m_beginColumn, ms->m_subcolumns,
            ms->m_beginRow, ms->m_subrows);
    printf("matrix == %llu %llu %llu\n", ms->m_matrix,
            ms->m_matrix->columns, ms->m_matrix->rows);
    printf("matrix1 == %llu %llu %llu \n", ms->m_matrix,
            ms->m_matrix->reValues, ms->m_matrix->imValues);
#endif
}

extern "C" __global__ void CUDAKernel_DotProductRe(
        MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_multiplyReMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProductIm(
        MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_multiplyImMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProduct(MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_multiplyMatrices(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_AddRe(
        MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_addReMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_AddIm(
        MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_addImMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_AddReIm(
        MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_addMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_SubstractRe(MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_substractReMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_SubstractIm(MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_substractImMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_Substract(MatrixStructure* output,
        MatrixStructure* params0, MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_substractMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_MultiplyConstantRe(MatrixStructure* output,
        MatrixStructure* params0, floatt* value) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_multiplyConstantReMatrix(output, params0, value,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_MultiplyConstantIm(MatrixStructure* output,
        MatrixStructure* params0, floatt* value) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_multiplyConstantImMatrix(output, params0, value,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_MultiplyConstant(MatrixStructure* output,
        MatrixStructure* params0, floatt* value) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_multiplyConstantMatrix(output, params0, value,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_TensorProductRe(MatrixStructure* output,
        MatrixStructure* params0,
        MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    UDA_tensorProductReMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_TensorProductIm(MatrixStructure* output, MatrixStructure* params0,
        MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_tensorProductImMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_TensorProduct(MatrixStructure* output,
        MatrixStructure* params0,
        MatrixStructure* params1) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_tensorProductMatrix(output, params0, params1,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_TransposeRe(MatrixStructure* output,
        MatrixStructure* params0) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_transposeReMatrix(output, params0,
            threadIndexX, threadIndexY);
}
    
extern "C" __global__ void CUDAKernel_TransposeIm(MatrixStructure* output,
        MatrixStructure* params0) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_transposeImMatrix(output, params0,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_Transpose(MatrixStructure* output,
        MatrixStructure* params0) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    CUDA_transposeMatrix(output, params0,
            threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_QRRe(MatrixStructure* output0,
        MatrixStructure* output1,
        MatrixStructure* params0) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
}

extern "C" __global__ void CUDAKernel_QRIm(MatrixStructure* output0,
        MatrixStructure* output1,
        MatrixStructure* params0) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
}

extern "C" __global__ void CUDAKernel_QR(MatrixStructure* output0,
        MatrixStructure* output1,
        MatrixStructure* params0) {
    uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
}
