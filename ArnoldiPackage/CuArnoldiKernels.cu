#include "hip/hip_runtime.h"
#include "CuArnodliProceduresImpl.h"
#include "CuMatrixUtils.h"
#include "Types.h"

extern "C" __global__ void CUDAKernel_CalculateTriangularH(math::Matrix* H,
        math::Matrix* Q, math::Matrix* R,
        math::Matrix* temp, math::Matrix* temp1,
        math::Matrix* temp2, math::Matrix* temp3,
        math::Matrix* temp4, math::Matrix* temp5) {
    CUDA_DEBUG();
    CUDA_CalculateTriangularH(H, Q, R, temp, temp1, temp2, temp3, temp4, temp5);
    CUDA_DEBUG();
}

#if 0

extern "C" __global__ void CUDAKernel_CalculateH(
        math::Matrix* H, math::Matrix* A,
        math::Matrix* w, math::Matrix* v,
        math::Matrix* f, math::Matrix* V, math::Matrix* transposeV,
        math::Matrix* s, math::Matrix* vs,
        math::Matrix* h, math::Matrix* vh) {
    CUDA_DEBUG();
    CUDA_CalculateH(true, 0, H, A,
            w, v,
            f, V, transposeV,
            s, vs, h, vh);
}
#endif