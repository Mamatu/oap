#include "hip/hip_runtime.h"
#include "CuMatrixProcedures.h"
#include "CuMatrixUtils.h"

extern "C" __global__ void CUDAKernel_DotProductRe(math::Matrix* output,
                                                   math::Matrix* params0,
                                                   math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  cuda_debug_function();
  CUDA_dotProductRe(output, params0, params1, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProductIm(math::Matrix* output,
                                                   math::Matrix* params0,
                                                   math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_dotProductIm(output, params0, params1, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProduct(math::Matrix* output,
                                                 math::Matrix* params0,
                                                 math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_dotProduct(output, params0, params1, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProductReExp(math::Matrix* output,
                                                      math::Matrix* params0,
                                                      math::Matrix* params1,
                                                      MatrixEx* matrixEx) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_dotProductReEx(output, params0, params1, *matrixEx, threadIndexX,
                      threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProductImExp(math::Matrix* output,
                                                      math::Matrix* params0,
                                                      math::Matrix* params1,
                                                      MatrixEx* matrixEx) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_dotProductImEx(output, params0, params1, *matrixEx, threadIndexX,
                      threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProductEx(math::Matrix* output,
                                                   math::Matrix* params0,
                                                   math::Matrix* params1,
                                                   MatrixEx* matrixEx) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_dotProductEx(output, params0, params1, *matrixEx, threadIndexX,
                    threadIndexY);
}

extern "C" __global__ void CUDAKernel_DotProductReOpt(math::Matrix* output,
                                                      math::Matrix* params0,
                                                      math::Matrix* params1) {
  extern __shared__ floatt bufferFloat[];
  CUDA_dotProductReOpt(output, params0, params1, bufferFloat);
}

extern "C" __global__ void CUDAKernel_DotProductImOpt(math::Matrix* output,
                                                      math::Matrix* params0,
                                                      math::Matrix* params1) {
  extern __shared__ floatt bufferFloat[];
  CUDA_dotProductImOpt(output, params0, params1, bufferFloat);
}

extern "C" __global__ void CUDAKernel_DotProductOpt(math::Matrix* output,
                                                    math::Matrix* params0,
                                                    math::Matrix* params1) {
  extern __shared__ floatt bufferFloat[];
  CUDA_dotProductOpt(output, params0, params1, bufferFloat);
}

extern "C" __global__ void CUDAKernel_DotProductReExpOpt(math::Matrix* output,
                                                         math::Matrix* params0,
                                                         math::Matrix* params1,
                                                         MatrixEx* matrixEx) {
  extern __shared__ floatt bufferFloat[];
  CUDA_dotProductReExOpt(output, params0, params1, *matrixEx, bufferFloat);
}

extern "C" __global__ void CUDAKernel_DotProductImExpOpt(math::Matrix* output,
                                                         math::Matrix* params0,
                                                         math::Matrix* params1,
                                                         MatrixEx* matrixEx) {
  extern __shared__ floatt bufferFloat[];
  CUDA_dotProductImExOpt(output, params0, params1, *matrixEx, bufferFloat);
}

extern "C" __global__ void CUDAKernel_DotProductExOpt(math::Matrix* output,
                                                      math::Matrix* params0,
                                                      math::Matrix* params1,
                                                      MatrixEx* matrixEx) {
  extern __shared__ floatt bufferFloat[];
  CUDA_dotProductExOpt(output, params0, params1, *matrixEx, bufferFloat);
}

extern "C" __global__ void CUDAKernel_AddRe(math::Matrix* output,
                                            math::Matrix* params0,
                                            math::Matrix* params1) {
  CUDA_addReMatrices(output, params0, params1);
}

extern "C" __global__ void CUDAKernel_AddIm(math::Matrix* output,
                                            math::Matrix* params0,
                                            math::Matrix* params1) {
  CUDA_addImMatrices(output, params0, params1);
}

extern "C" __global__ void CUDAKernel_Add(math::Matrix* output,
                                          math::Matrix* params0,
                                          math::Matrix* params1) {
  CUDA_addMatrix(output, params0, params1);
}

extern "C" __global__ void CUDAKernel_SubstractRe(math::Matrix* output,
                                                  math::Matrix* params0,
                                                  math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_substractReMatrices(output, params0, params1, threadIndexX,
                           threadIndexY);
}

extern "C" __global__ void CUDAKernel_SubstractIm(math::Matrix* output,
                                                  math::Matrix* params0,
                                                  math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_substractImMatrices(output, params0, params1, threadIndexX,
                           threadIndexY);
}

extern "C" __global__ void CUDAKernel_SubstractReal(math::Matrix* output,
                                                    math::Matrix* params0,
                                                    math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_substractRealMatrices(output, params0, params1, threadIndexX,
                             threadIndexY);
}

extern "C" __global__ void CUDAKernel_Substract(math::Matrix* output,
                                                math::Matrix* params0,
                                                math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_substractMatrices(output, params0, params1, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_MultiplyConstantRe(math::Matrix* output,
                                                         math::Matrix* params0,
                                                         floatt re) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_multiplyConstantReMatrix(output, params0, re, threadIndexX,
                                threadIndexY);
}

extern "C" __global__ void CUDAKernel_MultiplyConstantIm(math::Matrix* output,
                                                         math::Matrix* params0,
                                                         floatt im) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_multiplyConstantImMatrix(output, params0, im, threadIndexX,
                                threadIndexY);
}

extern "C" __global__ void CUDAKernel_MultiplyConstant(math::Matrix* output,
                                                       math::Matrix* params0,
                                                       floatt re, floatt im) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_multiplyConstantMatrix(output, params0, re, im, threadIndexX,
                              threadIndexY);
}

extern "C" __global__ void CUDAKernel_MultiplyConstantReal(
    math::Matrix* output, math::Matrix* params0, floatt re, floatt im) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_multiplyConstantRealMatrix(output, params0, re, im, threadIndexX,
                                  threadIndexY);
}

extern "C" __global__ void CUDAKernel_TensorProductRe(math::Matrix* output,
                                                      math::Matrix* params0,
                                                      math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_tensorProductReMatrix(output, params0, params1, threadIndexX,
                             threadIndexY);
}

extern "C" __global__ void CUDAKernel_TensorProductIm(math::Matrix* output,
                                                      math::Matrix* params0,
                                                      math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_tensorProductImMatrix(output, params0, params1, threadIndexX,
                             threadIndexY);
}

extern "C" __global__ void CUDAKernel_TensorProduct(math::Matrix* output,
                                                    math::Matrix* params0,
                                                    math::Matrix* params1) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_tensorProductMatrix(output, params0, params1, threadIndexX,
                           threadIndexY);
}

extern "C" __global__ void CUDAKernel_TransposeRe(math::Matrix* output,
                                                  math::Matrix* params0) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_transposeReMatrix(output, params0, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_TransposeIm(math::Matrix* output,
                                                  math::Matrix* params0) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_transposeImMatrix(output, params0, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_Transpose(math::Matrix* output,
                                                math::Matrix* params0) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_transposeMatrix(output, params0, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_TransposeEx(math::Matrix* output,
                                                  math::Matrix* params0,
                                                  MatrixEx* matrixEx) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_transposeMatrixEx(output, params0, *matrixEx, threadIndexX,
                         threadIndexY);
}

extern "C" __global__ void CUDAKernel_Magnitude(floatt* value,
                                                math::Matrix* params0,
                                                floatt* buffer) {
  CUDA_magnitudeOpt(value, params0, buffer);
}

extern "C" __global__ void CUDAKernel_QRGRRe(
    math::Matrix* output0, math::Matrix* output1, math::Matrix* params0,
    math::Matrix* aux0, math::Matrix* aux1, math::Matrix* aux2,
    math::Matrix* aux3) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  // CUDA_QRRe(output0, output1, params0, aux0, aux1, aux2, aux3,
  //      threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_QRGRIm(
    math::Matrix* output0, math::Matrix* output1, math::Matrix* params0,
    math::Matrix* aux0, math::Matrix* aux1, math::Matrix* aux2,
    math::Matrix* aux3) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  // CUDA_QRIm(output0, output1, params0, aux0, aux1, aux2, aux3,
  //        threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_QRGR(
    math::Matrix* output0, math::Matrix* output1, math::Matrix* params0,
    math::Matrix* aux0, math::Matrix* aux1, math::Matrix* aux2,
    math::Matrix* aux3) {
  CUDA_QRGR(output0, output1, params0, aux0, aux1, aux2, aux3);
}

extern "C" __global__ void CUDAKernel_SetVector(math::Matrix* output,
                                                uintt index,
                                                math::Matrix* params0,
                                                uintt length) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_setVector(output, index, params0, length, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_GetVector(math::Matrix* output,
                                                uintt length,
                                                math::Matrix* params0,
                                                uintt index) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_getVector(output, length, params0, index, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_SetIdentity(math::Matrix* matrix) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_setIdentityMatrix(matrix, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_SetDiagonal(math::Matrix* matrix,
                                                  floatt re, floatt im) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_setDiagonalMatrix(matrix, re, im, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_Invert(math::Matrix* output,
                                             math::Matrix* matrix,
                                             math::Matrix* aux1,
                                             math::Matrix* aux2,
                                             math::Matrix* aux3) {
  CUDA_invertMatrix(output, matrix, aux1, aux2, aux3);
}

extern "C" __global__ void CUDAKernel_CompareRe(int* sums,
                                                math::Matrix* matrix1,
                                                math::Matrix* matrix2,
                                                int* buffer,
                                                uintt bufferLength) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_compareReMatrix(sums, matrix1, matrix2, buffer, threadIndexX,
                       threadIndexY);
}

extern "C" __global__ void CUDAKernel_Compare(int* sums, math::Matrix* matrix1,
                                              math::Matrix* matrix2,
                                              int* buffer, uintt bufferLength) {
  uintt threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
  uintt threadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
  CUDA_compare(sums, matrix1, matrix2, buffer, threadIndexX, threadIndexY);
}

extern "C" __global__ void CUDAKernel_CompareOpt(int* sums,
                                                 math::Matrix* matrix1,
                                                 math::Matrix* matrix2) {
  extern __shared__ int sharedBufferInt[];
  CUDA_compareOpt(sums, matrix1, matrix2, sharedBufferInt);
}

extern "C" __global__ void CUDAKernel_CompareOptVer2(int* sums,
                                                     math::Matrix* matrix1,
                                                     math::Matrix* matrix2) {
  extern __shared__ int sharedBufferInt[];
  CUDA_compareOptVer2(sums, matrix1, matrix2, sharedBufferInt);
}

extern "C" __global__ void CUDAKernel_MagnitudeOpt(floatt* sums,
                                                   math::Matrix* matrix) {
  extern __shared__ floatt bufferFloat[];
  CUDA_magnitudeOpt(sums, matrix, bufferFloat);
}

extern "C" __global__ void CUDAKernel_MagnitudeOptVer2(floatt* sums,
                                                       math::Matrix* matrix) {
  extern __shared__ floatt bufferFloat[];
  CUDA_magnitudeOptVer2(sums, matrix, bufferFloat);
}

extern "C" __global__ void CUDAKernel_IsUpperTriangular(int* outcome,
                                                        math::Matrix* matrix) {
  int is = CUDA_isUpperTriangular(matrix);
  (*outcome) = is;
}

extern "C" __global__ void CUDAKernel_CalculateTriangularH(
    math::Matrix* H, math::Matrix* Q, math::Matrix* R, math::Matrix* temp,
    math::Matrix* temp1, math::Matrix* temp2, math::Matrix* temp3,
    math::Matrix* temp4, math::Matrix* temp5) {
  CUDA_HMtoUTM(H, Q, R, temp, temp1, temp2, temp3, temp4, temp5);
}
